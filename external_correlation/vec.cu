#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cupti.h>
#include <stdio.h>
#include <string.h>
#include <omp.h>
#include <atomic>
#include <string>

#define DRIVER_API_CALL(apiFuncCall)                                           \
  do {                                                                           \
    hipError_t _status = apiFuncCall;                                            \
    if (_status != hipSuccess) {                                             \
      fprintf(stderr, "%s:%d: error: function %s failed with error %d.\n",   \
        __FILE__, __LINE__, #apiFuncCall, _status);                    \
      exit(-1);                                                              \
    }                                                                          \
  } while (0)

#define RUNTIME_API_CALL(apiFuncCall)                                          \
  do {                                                                           \
    hipError_t _status = apiFuncCall;                                         \
    if (_status != hipSuccess) {                                              \
      fprintf(stderr, "%s:%d: error: function %s failed with error %s.\n",   \
        __FILE__, __LINE__, #apiFuncCall, hipGetErrorString(_status));\
      exit(-1);                                                              \
    }                                                                          \
  } while (0)

#define COMPUTE_N 50000

extern void initTrace(void);
extern void finiTrace(void);

extern __thread int64_t localId;

static std::atomic<int> correlationId;


static __global__ void
vecAdd(const int* A, const int* B, int* C, int N)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < N)
    C[i] = A[i] + B[i];
}


static __global__ void
vecSub(const int* A, const int* B, int* C, int N)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < N)
    C[i] = A[i] - B[i];
}


static void
do_pass_runtime(hipDevice_t device)
{
  int *h_A, *h_B, *h_C;
  int *d_A1, *d_B1, *d_C1;
  int *d_A2, *d_B2, *d_C2;
  size_t size = COMPUTE_N * sizeof(int);
  int threadsPerBlock = 256;
  int blocksPerGrid = 0;

  // Allocate input vectors h_A and h_B in host memory
  // don't bother to initialize
  h_A = (int*)malloc(size);
  h_B = (int*)malloc(size);
  h_C = (int*)malloc(size);

  // Allocate vectors in device memory
  RUNTIME_API_CALL(hipMalloc((void**)&d_A1, size));
  RUNTIME_API_CALL(hipMalloc((void**)&d_B1, size));
  RUNTIME_API_CALL(hipMalloc((void**)&d_C1, size));
  RUNTIME_API_CALL(hipMemcpy(d_A1, h_A, size, hipMemcpyHostToDevice));
  RUNTIME_API_CALL(hipMemcpy(d_B1, h_B, size, hipMemcpyHostToDevice));

  RUNTIME_API_CALL(hipMalloc((void**)&d_A2, size));
  RUNTIME_API_CALL(hipMalloc((void**)&d_B2, size));
  RUNTIME_API_CALL(hipMalloc((void**)&d_C2, size));
  RUNTIME_API_CALL(hipMemcpy(d_A2, h_A, size, hipMemcpyHostToDevice));
  RUNTIME_API_CALL(hipMemcpy(d_B2, h_B, size, hipMemcpyHostToDevice));

  blocksPerGrid = (COMPUTE_N + threadsPerBlock - 1) / threadsPerBlock;

  // Launch kernel
  localId = correlationId.fetch_add(1);
  printf("Run VecAdd\n");
  vecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A1, d_B1, d_C1, COMPUTE_N);

  localId = correlationId.fetch_add(1);
  printf("Run VecSub\n");
  vecSub<<<blocksPerGrid, threadsPerBlock>>>(d_A2, d_B2, d_C2, COMPUTE_N);

  // Clean up
  RUNTIME_API_CALL(hipFree(d_A1));
  RUNTIME_API_CALL(hipFree(d_B1));
  RUNTIME_API_CALL(hipFree(d_C1));
  RUNTIME_API_CALL(hipFree(d_A2));
  RUNTIME_API_CALL(hipFree(d_B2));
  RUNTIME_API_CALL(hipFree(d_C2));

  free(h_A);
  free(h_B);
  free(h_C);
}


static void
do_pass_same_context(hipDevice_t device)
{
  hipCtx_t deviceContext;
  hipCtxCreate(&deviceContext, 0, device);
  CUpti_ActivityPCSamplingConfig configPC;
  configPC.samplingPeriod = CUPTI_ACTIVITY_PC_SAMPLING_PERIOD_MIN;
  cuptiActivityConfigurePCSampling(deviceContext, &configPC);

  hipModule_t moduleAdd;
  hipFunction_t vecAdd;
  DRIVER_API_CALL(hipModuleLoad(&moduleAdd, "vecAdd.cubin"));
  DRIVER_API_CALL(hipModuleGetFunction(&vecAdd, moduleAdd, "vecAdd"));

  hipModule_t moduleSub;
  hipFunction_t vecSub;
  DRIVER_API_CALL(hipModuleLoad(&moduleSub, "vecSub.cubin"));
  DRIVER_API_CALL(hipModuleGetFunction(&vecSub, moduleSub, "vecSub"));

#pragma omp parallel 
  {
    hipCtx_t context;
    hipCtxSetCurrent(deviceContext);
    hipCtxGetCurrent(&context);

    int *h_A, *h_B, *h_C;
    int *d_A1, *d_B1, *d_C1;
    int *d_A2, *d_B2, *d_C2;
    size_t size = COMPUTE_N * sizeof(int);
    int threadsPerBlock = 256;
    int blocksPerGrid = 0;

    // Allocate input vectors h_A and h_B in host memory
    // don't bother to initialize
    h_A = (int*)malloc(size);
    h_B = (int*)malloc(size);
    h_C = (int*)malloc(size);

    blocksPerGrid = (COMPUTE_N + threadsPerBlock - 1) / threadsPerBlock;
    int computeN = COMPUTE_N;

    if (omp_get_thread_num() == 0) {
      // Allocate vectors in device memory
      RUNTIME_API_CALL(hipMalloc((void**)&d_A1, size));
      RUNTIME_API_CALL(hipMalloc((void**)&d_B1, size));
      RUNTIME_API_CALL(hipMalloc((void**)&d_C1, size));
      RUNTIME_API_CALL(hipMemcpy(d_A1, h_A, size, hipMemcpyHostToDevice));
      RUNTIME_API_CALL(hipMemcpy(d_B1, h_B, size, hipMemcpyHostToDevice));

      // Launch kernel
      localId = correlationId.fetch_add(1);
      printf("Run VecAdd\n");
      void *args1[4] = {
        &d_A1, &d_B1, &d_C1, &computeN
      };
      DRIVER_API_CALL(hipModuleLaunchKernel(vecAdd, blocksPerGrid, 1, 1, threadsPerBlock, 1, 1, 0, 0, args1, 0));

      // Clean up
      RUNTIME_API_CALL(hipFree(d_A1));
      RUNTIME_API_CALL(hipFree(d_B1));
      RUNTIME_API_CALL(hipFree(d_C1));
    } else if (omp_get_thread_num() == 1) {
      // Allocate vectors in device memory
      RUNTIME_API_CALL(hipMalloc((void**)&d_A2, size));
      RUNTIME_API_CALL(hipMalloc((void**)&d_B2, size));
      RUNTIME_API_CALL(hipMalloc((void**)&d_C2, size));
      RUNTIME_API_CALL(hipMemcpy(d_A2, h_A, size, hipMemcpyHostToDevice));
      RUNTIME_API_CALL(hipMemcpy(d_B2, h_B, size, hipMemcpyHostToDevice));

      // Launch kernel
      localId = correlationId.fetch_add(1);
      printf("Run VecSub\n");
      void *args2[4] = {
        &d_A2, &d_B2, &d_C2, &computeN
      };
      DRIVER_API_CALL(hipModuleLaunchKernel(vecSub, blocksPerGrid, 1, 1, threadsPerBlock, 1, 1, 0, 0, args2, 0));

      // Clean up
      RUNTIME_API_CALL(hipFree(d_A2));
      RUNTIME_API_CALL(hipFree(d_B2));
      RUNTIME_API_CALL(hipFree(d_C2));
    }
    free(h_A);
    free(h_B);
    free(h_C);
  }
}


static void
do_pass_diff_context(hipDevice_t device)
{
#pragma omp parallel 
  {
    hipCtx_t context;
    hipCtxCreate(&context, 0, device);
    CUpti_ActivityPCSamplingConfig configPC;
    configPC.samplingPeriod = CUPTI_ACTIVITY_PC_SAMPLING_PERIOD_MIN;
    cuptiActivityConfigurePCSampling(context, &configPC);

    int *h_A, *h_B, *h_C;
    int *d_A1, *d_B1, *d_C1;
    int *d_A2, *d_B2, *d_C2;
    size_t size = COMPUTE_N * sizeof(int);
    int threadsPerBlock = 256;
    int blocksPerGrid = 0;

    // Allocate input vectors h_A and h_B in host memory
    // don't bother to initialize
    h_A = (int*)malloc(size);
    h_B = (int*)malloc(size);
    h_C = (int*)malloc(size);

    blocksPerGrid = (COMPUTE_N + threadsPerBlock - 1) / threadsPerBlock;
    int computeN = COMPUTE_N;

    if (omp_get_thread_num() == 0) {
      hipModule_t moduleAdd;
      hipFunction_t vecAdd;
      DRIVER_API_CALL(hipModuleLoad(&moduleAdd, "vecAdd.cubin"));
      DRIVER_API_CALL(hipModuleGetFunction(&vecAdd, moduleAdd, "vecAdd"));

      // Allocate vectors in device memory
      RUNTIME_API_CALL(hipMalloc((void**)&d_A1, size));
      RUNTIME_API_CALL(hipMalloc((void**)&d_B1, size));
      RUNTIME_API_CALL(hipMalloc((void**)&d_C1, size));
      RUNTIME_API_CALL(hipMemcpy(d_A1, h_A, size, hipMemcpyHostToDevice));
      RUNTIME_API_CALL(hipMemcpy(d_B1, h_B, size, hipMemcpyHostToDevice));

      // Launch kernel
      localId = correlationId.fetch_add(1);
      printf("Run VecAdd\n");
      void *args1[4] = {
        &d_A1, &d_B1, &d_C1, &computeN
      };
      DRIVER_API_CALL(hipModuleLaunchKernel(vecAdd, blocksPerGrid, 1, 1, threadsPerBlock, 1, 1, 0, 0, args1, 0));

      // Clean up
      RUNTIME_API_CALL(hipFree(d_A1));
      RUNTIME_API_CALL(hipFree(d_B1));
      RUNTIME_API_CALL(hipFree(d_C1));
    } else if (omp_get_thread_num() == 1) {
      hipModule_t moduleSub;
      hipFunction_t vecSub;
      DRIVER_API_CALL(hipModuleLoad(&moduleSub, "vecSub.cubin"));
      DRIVER_API_CALL(hipModuleGetFunction(&vecSub, moduleSub, "vecSub"));

      // Allocate vectors in device memory
      RUNTIME_API_CALL(hipMalloc((void**)&d_A2, size));
      RUNTIME_API_CALL(hipMalloc((void**)&d_B2, size));
      RUNTIME_API_CALL(hipMalloc((void**)&d_C2, size));
      RUNTIME_API_CALL(hipMemcpy(d_A2, h_A, size, hipMemcpyHostToDevice));
      RUNTIME_API_CALL(hipMemcpy(d_B2, h_B, size, hipMemcpyHostToDevice));

      // Launch kernel
      localId = correlationId.fetch_add(1);
      printf("Run VecSub\n");
      void *args2[4] = {
        &d_A2, &d_B2, &d_C2, &computeN
      };
      DRIVER_API_CALL(hipModuleLaunchKernel(vecSub, blocksPerGrid, 1, 1, threadsPerBlock, 1, 1, 0, 0, args2, 0));

      // Clean up
      RUNTIME_API_CALL(hipFree(d_A2));
      RUNTIME_API_CALL(hipFree(d_B2));
      RUNTIME_API_CALL(hipFree(d_C2));
    }
    free(h_A);
    free(h_B);
    free(h_C);
  }
}


int
main(int argc, char *argv[])
{
  if (argc < 2) {
    exit(-1);
  }
  std::string option = std::string(argv[1]);

  hipDevice_t device;  
  char deviceName[32];
  int deviceNum = 0, devCount = 0;

  // initialize the activity trace
  // make sure activity is enabled before any CUDA API
  initTrace();

  DRIVER_API_CALL(hipInit(0));

  RUNTIME_API_CALL(hipGetDeviceCount(&devCount));
  for (deviceNum=0; deviceNum<devCount; deviceNum++) {
    DRIVER_API_CALL(hipDeviceGet(&device, deviceNum));
    DRIVER_API_CALL(hipDeviceGetName(deviceName, 32, device));
    printf("Device Name: %s\n", deviceName);

    RUNTIME_API_CALL(hipSetDevice(deviceNum));

    if (option == "runtime") {
      do_pass_runtime(device);
    } else if (option == "same_context") {
      do_pass_same_context(device);
    } else if (option == "diff_context") {
      do_pass_diff_context(device);
    } else {
      exit(-1);
    }

    RUNTIME_API_CALL(hipDeviceSynchronize());
    RUNTIME_API_CALL(hipDeviceReset());

    finiTrace();
  }
  return 0;
}
