
#include <hip/hip_runtime.h>
extern "C"
__global__ void vecAdd(const int* A, const int* B, int* C, int N)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < N)
    C[i] = A[i] + B[i];
}
